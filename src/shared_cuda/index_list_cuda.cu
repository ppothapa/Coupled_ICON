#include "index_list_cuda.h"

#ifndef __HIP__
#include <hip/hip_runtime.h>
#include <cub/device/device_select.cuh>
#include <cub/iterator/counting_input_iterator.cuh>

template<typename T>
struct ZeroCmp
{
        const T* conditions;
        const int startid;

        ZeroCmp(const int startid, const T* conditions) :
                startid(startid), conditions(conditions)
        { }

        __device__ __host__ __forceinline__
        bool operator() (const int &id)
        {
          return (conditions[ id - startid ] != 0);
        }
};

template <typename T>
static
void c_generate_index_list_generic_device(
                        const T* dev_conditions,
                        const int startid, const int endid,
                        int* dev_indices,
                        int* dev_nvalid, hipStream_t stream)
{
        static size_t storageSize = 0;
        static char* storage = nullptr;

        const int n = endid - startid + 1;

        // Argument is the offset of the first element
        hipcub::CountingInputIterator<int> iterator(startid);

        // Determine temporary device storage requirements
        size_t storageRequirement;
        hipcub::DeviceSelect::Flagged(nullptr, storageRequirement,
                        iterator, dev_conditions, dev_indices,
                        dev_nvalid, n, stream);

        // Allocate temporary storage (only if not enough)
        if (storageRequirement > storageSize)
        {
                hipFree(storage);
                hipMalloc(&storage, storageRequirement);
                storageSize = storageRequirement;
        }

        ZeroCmp<T> select(startid, dev_conditions);
        hipcub::DeviceSelect::If(storage, storageRequirement,
                        iterator, dev_indices,
                        dev_nvalid, n,
                        select, stream);
}


template <typename T>
static
void c_generate_index_list_batched_generic(
                        const int batch_size,
                        const T* dev_conditions, const int cond_stride,
                        const int startid, const int endid,
                        int* dev_indices, const int idx_stride,
                        int* dev_nvalid, hipStream_t stream)
{
        for (int i = 0; i < batch_size; i++)
                c_generate_index_list_generic_device(
                                dev_conditions + cond_stride*i,
                                startid, endid,
                                dev_indices + idx_stride*i,
                                dev_nvalid + i, stream);
}

template <typename T>
static
void c_generate_index_list_generic(
                        const T* dev_conditions,
                        const int startid, const int endid,
                        int* dev_indices,
                        int& nvalid, hipStream_t stream)
{
        static int* dev_nvalid = nullptr;
        if (dev_nvalid == nullptr)
                        hipMalloc(&dev_nvalid, sizeof(int));

        c_generate_index_list_generic_device(
                        dev_conditions, startid, endid, dev_indices, dev_nvalid, stream);

        hipMemcpyAsync(&nvalid, dev_nvalid, sizeof(int), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);
}

///
/// Exposed functions
/// 
/// Non-batched first
/// 

void c_generate_index_list_i1(
                        const char* dev_conditions,
                        const int startid, const int endid,
                        int* dev_indices,
                        int& nvalid, hipStream_t stream)
{
        c_generate_index_list_generic(dev_conditions, startid, endid, dev_indices, nvalid, stream);
}

void c_generate_index_list_i4(
                        const int* dev_conditions,
                        const int startid, const int endid,
                        int* dev_indices,
                        int& nvalid, hipStream_t stream)
{
        c_generate_index_list_generic(dev_conditions, startid, endid, dev_indices, nvalid, stream);
}

/// 
/// And now batched
/// 

void c_generate_index_list_batched_i1(
        const int batch_size,
        const char* dev_conditions, const int cond_stride,
        const int startid, const int endid,
        int* dev_indices, const int idx_stride,
        int* dev_nvalid, hipStream_t stream)
{
c_generate_index_list_batched_generic(
                batch_size,
                dev_conditions, cond_stride,
                startid, endid,
                dev_indices, idx_stride,
                dev_nvalid, stream);
}

void c_generate_index_list_batched_i4(
                const int batch_size,
                const int* dev_conditions, const int cond_stride,
                const int startid, const int endid,
                int* dev_indices, const int idx_stride,
                int* dev_nvalid, hipStream_t stream)
{
        c_generate_index_list_batched_generic(
                        batch_size,
                        dev_conditions, cond_stride,
                        startid, endid,
                        dev_indices, idx_stride,
                        dev_nvalid, stream);
}

#else

// HIP implementation

#include <hip/hip_runtime.h>
#include <hipcub/device/device_select.hpp>
#include <hipcub/iterator/counting_input_iterator.hpp>

template<typename T>
struct ZeroCmp
{
	const T* conditions;
	const int startid;

	ZeroCmp(const int startid, const T* conditions) :
		startid(startid), conditions(conditions)
	{ }

	__device__ __host__ __forceinline__
	bool operator() (const int &id)
	{
	  return (conditions[ id - startid ] != 0);
	}
};

template <typename T>
static
void c_generate_index_list_generic_device(
			const T* dev_conditions,
			const int startid, const int endid,
			int* dev_indices,
			int* dev_nvalid, hipStream_t stream)
{
	static size_t storageSize = 0;
	static char* storage = nullptr;

	const int n = endid - startid + 1;

	// Argument is the offset of the first element
	hipcub::CountingInputIterator<int> iterator(startid);

	// Determine temporary device storage requirements
	size_t storageRequirement;
	hipcub::DeviceSelect::Flagged(nullptr, storageRequirement,
			iterator, dev_conditions, dev_indices,
			dev_nvalid, n, 0);

	// Allocate temporary storage (only if not enough)
	if (storageRequirement > storageSize)
	{
		hipFree(storage);
		hipMalloc(&storage, storageRequirement);
		storageSize = storageRequirement;
	}

	ZeroCmp<T> select(startid, dev_conditions);
	hipcub::DeviceSelect::If(storage, storageRequirement,
			iterator, dev_indices,
			dev_nvalid, n,
			select, 0);
}


template <typename T>
static
void c_generate_index_list_batched_generic(
			const int batch_size,
			const T* dev_conditions, const int cond_stride,
			const int startid, const int endid,
			int* dev_indices, const int idx_stride,
			int* dev_nvalid, hipStream_t stream)
{
	for (int i = 0; i < batch_size; i++)
		c_generate_index_list_generic_device(
				dev_conditions + cond_stride*i,
				startid, endid,
				dev_indices + idx_stride*i,
				dev_nvalid + i, 0);
}

template <typename T>
static
void c_generate_index_list_generic(
			const T* dev_conditions,
			const int startid, const int endid,
			int* dev_indices,
			int& nvalid, hipStream_t stream)
{
	static int* dev_nvalid = nullptr;
	if (dev_nvalid == nullptr)
			hipMalloc(&dev_nvalid, sizeof(int));

	c_generate_index_list_generic_device(
			dev_conditions, startid, endid, dev_indices, dev_nvalid, 0);

	hipMemcpyAsync(&nvalid, dev_nvalid, sizeof(int), hipMemcpyDeviceToHost, 0);
	hipStreamSynchronize(0);
}

///
/// Exposed functions
/// 
/// Non-batched first
/// 

void c_generate_index_list_i1(
			const char* dev_conditions,
			const int startid, const int endid,
			int* dev_indices,
			int& nvalid, hipStream_t stream)
{
	c_generate_index_list_generic(dev_conditions, startid, endid, dev_indices, nvalid, 0);
}

void c_generate_index_list_i4(
			const int* dev_conditions,
			const int startid, const int endid,
			int* dev_indices,
			int& nvalid, hipStream_t stream)
{
	c_generate_index_list_generic(dev_conditions, startid, endid, dev_indices, nvalid, 0);
}

/// 
/// And now batched
/// 

void c_generate_index_list_batched_i1(
	const int batch_size,
	const char* dev_conditions, const int cond_stride,
	const int startid, const int endid,
	int* dev_indices, const int idx_stride,
	int* dev_nvalid, hipStream_t stream)
{
c_generate_index_list_batched_generic(
		batch_size,
		dev_conditions, cond_stride,
		startid, endid,
		dev_indices, idx_stride,
		dev_nvalid, 0);
}

void c_generate_index_list_batched_i4(
		const int batch_size,
		const int* dev_conditions, const int cond_stride,
		const int startid, const int endid,
		int* dev_indices, const int idx_stride,
		int* dev_nvalid, hipStream_t stream)
{
	c_generate_index_list_batched_generic(
			batch_size,
			dev_conditions, cond_stride,
			startid, endid,
			dev_indices, idx_stride,
			dev_nvalid, 0);
}

void initHIP(int deviceNum){
	hipSetDevice(deviceNum);
}
#endif 
